#include "hip/hip_runtime.h"
/**
 * Given a number of routers N, a number of links R, and a NxN traffic matrix TM
 * where TM[i][j] is the number of bytes sent from router i to router j,
 * Find G, the topology with N nodes and R edges such that the cost of sending
 * the data in TM is minimized, where the cost is defined as the total number
 * of bytes sent by all routers when traffic is routed by a shortest-path
 * policy.
 */

#include <fstream>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/transform_reduce.h>

#define MAX_GRAPHS (1 << 28)
#define THREADS_PER_BLOCK 256

// To allow static allocation of data structures on the stack of GPU threads, we
// set a constant MAX_NODES that this program accepts here.
#define MAX_NODES 16
#define MAX_EDGES (MAX_NODES * MAX_NODES)

#define CuAlloc(sz)                                                            \
  ({                                                                           \
    void *tmp = NULL;                                                          \
    auto err = hipMalloc(&tmp, sz);                                           \
    if (tmp == NULL) {                                                         \
      fprintf(stderr, "Failed to allocate device vector!\n");                  \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
    tmp;                                                                       \
  })

#define min(a, b)                                                              \
  ({                                                                           \
    auto tmpa = (a);                                                           \
    auto tmpb = (b);                                                           \
    tmpa < tmpb ? tmpa : tmpb;                                                 \
  })

#define max(a, b)                                                              \
  ({                                                                           \
    auto tmpa = (a);                                                           \
    auto tmpb = (b);                                                           \
    tmpa > tmpb ? tmpa : tmpb;                                                 \
  })

// Tuple of graph id and cost, this can't just be a std::pair because std::pair
// isn't defined with __device__.
struct Pair {
  size_t graph;
  size_t cost;

  __host__ __device__ Pair(size_t graph, size_t cost)
      : graph(graph), cost(cost) {}

  __host__ __device__ Pair() : graph(0), cost(0) {}

  __host__ __device__ Pair operator=(Pair other) {
    this->graph = other.graph;
    this->cost = other.cost;
    return *this;
  }
};

__host__ __device__ size_t choose(size_t n, size_t r) {
  size_t a = min(n - r, r);
  size_t b = max(n - r, r);

  size_t num = 1;
  for (size_t i = 0; i < a; i++) {
    num *= (n - i);
  }

  size_t denom = 1;
  for (size_t i = 1; i <= a; i++) {
    denom *= i;
  }

  return num / denom;
}

/**
https://stackoverflow.com/a/57076790

def combination_at_idx(idx, elems, r):
    if len(elems) == r:
        # We are looking for r elements in a list of size r - thus, we need
        # each element.
        return elems

    if len(elems) == 0 or len(elems) < r:
        return []

    combinations = choose(len(elems), r)    # total number of combinations
    remains = choose(len(elems) - 1, r)     # combinations after selection

    offset = combinations - remains

    if idx >= offset:       # combination does not start with first element
        return combination_at_idx(idx - offset, elems[1:], r)

    # We now know the first element of the combination, but *not* yet the next
    # r - 1 elements. These need to be computed as well, again recursively.
    return [elems[0]] + combination_at_idx(idx, elems[1:], r - 1)
*/

__host__ __device__ void combination_at_idx(size_t idx, size_t *elems_out,
                                            size_t first_elem, size_t n,
                                            size_t r) {
  if (first_elem == (n - r)) {
    for (size_t i = 0; i < r; i++) {
      elems_out[i] = first_elem + i;
    }
  }

  if (first_elem >= (n - r)) {
    return;
  }

  size_t combos = choose(n - first_elem, r);
  size_t remains = choose(n - first_elem - 1, r);

  size_t offset = combos - remains;
  if (idx >= offset) {
    return combination_at_idx(idx - offset, elems_out, first_elem + 1, n, r);
  }

  elems_out[0] = first_elem;
  return combination_at_idx(idx, elems_out + 1, first_elem + 1, n, r - 1);
}

__device__ bool contains(size_t *bag, size_t len, size_t elem) {
  for (size_t i = 0; i < len; i++) {
    if (bag[i] == elem) {
      return true;
    }
  }
  return false;
}

__device__ size_t n_connected(size_t root, size_t n_nodes, size_t n_edges,
                              size_t *srcs, size_t *dsts) {
  // count how many nodes a DFS rooted at `root` reaches
  size_t frontier[MAX_EDGES];
  size_t frontier_len = root;
  frontier[frontier_len++] = 0;

  size_t visited[MAX_NODES];
  size_t visited_len = 0;

  while (frontier_len > 0) {
    size_t nid = frontier[--frontier_len];

    if (contains(visited, visited_len, nid)) {
      continue;
    }
    visited[visited_len++] = nid;

    for (size_t i = 0; i < n_edges; i++) {
      if (srcs[i] == nid) {
        if (!contains(visited, visited_len, dsts[i])) {
          frontier[frontier_len++] = dsts[i];
        }
      } else if (dsts[i] == nid) {
        if (!contains(visited, visited_len, srcs[i])) {
          frontier[frontier_len++] = srcs[i];
        }
      }
    }
  }

  return visited_len;
}

/**
 * Returns true if the graph described by the src/dst edge pairs is fully
 * connected, false otherwise
 */
__device__ bool is_connected(size_t n_nodes, size_t n_edges, size_t *srcs,
                             size_t *dsts) {
  return n_connected(0, n_nodes, n_edges, srcs, dsts) == n_nodes;
}

__global__ void project_tm_to_graph(size_t offset, Pair *out,
                                    size_t *traffix_matrix, size_t *elist_src,
                                    size_t *elist_dst, size_t n_graphs,
                                    size_t n_nodes, size_t n_edges) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i > n_graphs) {
    // This index is invalid, and doesn't correspond to a valid graph
    return;
  }

  size_t n_total_edges = n_nodes * (n_nodes - 1) / 2;

  // Get the i'th combination of selecting `n_edges` edges from all possible
  // edges.
  size_t g[MAX_EDGES];
  combination_at_idx(offset + i, g, 0, n_total_edges, n_edges);

  // Map the edge id to a src/dst pair
  size_t src[MAX_EDGES];
  size_t dst[MAX_EDGES];
  for (size_t eid = 0; eid < n_edges; eid++) {
    src[eid] = elist_src[g[eid]];
    dst[eid] = elist_dst[g[eid]];
  }

  if (!is_connected(n_nodes, n_edges, src, dst)) {
    out[i] = Pair(i, 0);
  } else {

    // Floyd Warshall Algorithm
    size_t min_dists[MAX_EDGES];

    for (size_t src_nid = 0; src_nid < n_nodes; src_nid++) {
      for (size_t dst_nid = 0; dst_nid < n_nodes; dst_nid++) {
        min_dists[src_nid * n_nodes + dst_nid] = n_edges + 1;
      }
    }

    for (size_t eid = 0; eid < n_edges; eid++) {
      size_t src_nid = src[eid];
      size_t dst_nid = dst[eid];

      min_dists[src_nid * n_nodes + dst_nid] = 1;
      min_dists[dst_nid * n_nodes + src_nid] = 1;
    }

    for (size_t i_node = 0; i_node < n_nodes; i_node++) {
      for (size_t src_nid = 0; src_nid < n_nodes; src_nid++) {
        for (size_t dst_nid = (src_nid + 1); dst_nid < n_nodes; dst_nid++) {
          min_dists[src_nid * n_nodes + dst_nid] =
              min(min_dists[src_nid * n_nodes + dst_nid],
                  min_dists[src_nid * n_nodes + i_node] +
                      min_dists[i_node * n_nodes + dst_nid]);
        }
      }
    }

    size_t cost = 0;
    for (size_t src_nid = 0; src_nid < n_nodes; src_nid++) {
      for (size_t dst_nid = src_nid + 1; dst_nid < n_nodes; dst_nid++) {
        auto n_hops = min_dists[src_nid * n_nodes + dst_nid];
        cost += n_hops * traffix_matrix[src_nid * n_nodes + dst_nid];
        cost += n_hops * traffix_matrix[dst_nid * n_nodes + src_nid];
      }
    }

    out[i] = Pair(offset + i, cost);
  }
}

__host__ std::vector<std::string> split(std::string s, std::string delimiter) {
  size_t pos_start = 0, pos_end, delim_len = delimiter.length();
  std::string token;
  std::vector<std::string> res;

  while ((pos_end = s.find(delimiter, pos_start)) != std::string::npos) {
    token = s.substr(pos_start, pos_end - pos_start);
    pos_start = pos_end + delim_len;
    res.push_back(token);
  }

  res.push_back(s.substr(pos_start));
  return res;
}

struct mingraph {
  using T = struct Pair;
  /*! \typedef first_argument_type
   *  \brief The type of the function object's first argument.
   */
  typedef T first_argument_type;

  /*! \typedef second_argument_type
   *  \brief The type of the function object's second argument.
   */
  typedef T second_argument_type;

  /*! \typedef result_type
   *  \brief The type of the function object's result;
   */
  typedef T result_type;

  __thrust_exec_check_disable__ __host__ __device__ struct Pair
  operator()(const T &lhs, const T &rhs) const {
    if (lhs.cost == 0) {
      return rhs;
    }

    if (rhs.cost == 0) {
      return lhs;
    }
    return (lhs.cost < rhs.cost) ? lhs : rhs;
  }
};

int main(int argc, char **argv) {
  hipError_t err;

  assert(argc >= 4);

  size_t n_nodes = atoi(argv[1]);
  size_t n_edges = atoi(argv[2]);
  size_t n_total_edges = n_nodes * (n_nodes - 1) / 2;
  // Connected graphs need at least N - 1 edges
  assert(n_edges > (n_nodes - 1));
  // Check that n_edges isn't more than the number of possible edges because we
  // aren't considering multigraphs
  assert(n_edges <= n_total_edges);
  assert(n_nodes <= MAX_NODES);

  char const *filename = argv[3];

  printf("n_nodes = %zu\n", n_nodes);
  printf("n_edges = %zu\n", n_edges);

  // Read the input matrix to be projected onto each graph
  std::vector<size_t> traffic_matrix(n_nodes * n_nodes);

  std::ifstream ifile(filename);
  std::string str;
  size_t src = 0;
  while (std::getline(ifile, str)) {
    auto parts = split(str, " ");
    for (size_t dst = 0; dst < parts.size(); dst++) {
      traffic_matrix[src * n_nodes + dst] +=
          strtoul(parts[dst].c_str(), NULL, 10);
    }
    src++;
  }

  // Copy traffic matrix to GPU
  size_t *tm = (size_t *)CuAlloc(sizeof(size_t) * n_nodes * n_nodes);
  err = hipMemcpy(tm, traffic_matrix.data(),
                   sizeof(size_t) * traffic_matrix.size(),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  // Count the total number of graphs with n_nodes nodes and n_edges edges
  printf("n_graphs = %zu C %zu\n", n_total_edges, n_edges);
  size_t n_graphs = choose(n_total_edges, n_edges);
  printf("n_graphs = %zu\n", n_graphs);

  // allocate output buffer (pair of graph-id, cost of projection) - for large
  // values of n_nodes we can't allocate a large enough buffer, so instead we
  // choose a fixed size output and process one chunk of graphs at a time.
  Pair *output = (Pair *)CuAlloc(sizeof(Pair) * MAX_GRAPHS);
  printf("Allocated output vector\n");

  // Create a mapping from edge ids in [0, n_total_edges) to (src, dst)
  // i.e. the edge with id eid's source and dest is given by srcs[eid],
  // dsts[eid] respectively.
  std::vector<size_t> srcs;
  std::vector<size_t> dsts;
  for (size_t i = 0; i < n_nodes; i++) {
    for (size_t j = (i + 1); j < n_nodes; j++) {
      srcs.push_back(i);
      dsts.push_back(j);
    }
  }
  auto dev_src_list = (size_t *)CuAlloc(sizeof(size_t) * srcs.size());
  printf("Allocated edge src list\n");
  auto dev_dst_list = (size_t *)CuAlloc(sizeof(size_t) * dsts.size());
  printf("Allocated edge dst list\n");

  err = hipMemcpy(dev_src_list, srcs.data(), sizeof(size_t) * srcs.size(),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(dev_dst_list, dsts.data(), sizeof(size_t) * dsts.size(),
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    exit(EXIT_FAILURE);
  }

  // Comparator for finding the minimum element in output
  auto min_fn = mingraph();

  // Min element so far
  Pair acc = Pair();

  for (size_t offset = 0; offset < n_graphs; offset += MAX_GRAPHS) {
    // Number of graphs processed by this iteration
    size_t n_graphs_iter = min(MAX_GRAPHS, n_graphs - offset);

    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (n_graphs_iter + threadsPerBlock - 1) / threadsPerBlock;
    project_tm_to_graph<<<blocksPerGrid, threadsPerBlock>>>(
        offset, output, tm, dev_src_list, dev_dst_list, n_graphs, n_nodes,
        n_edges);

    // Wrap raw ptr with a device_ptr so that we can call thrust::reduce below
    thrust::device_ptr<Pair> dev_ptr = thrust::device_pointer_cast(output);

    // Get the minimum graph-id and the associated cost for the current chunk
    auto curr_min =
        thrust::reduce(dev_ptr, dev_ptr + n_graphs_iter, Pair(0, 0), min_fn);

    // Copy results to host (for debugging)
    // size_t *host_output = (size_t *)malloc(sizeof(size_t) * n_graphs);
    // err = hipMemcpy(host_output, output, sizeof(size_t) * n_graphs,
    //                  hipMemcpyDeviceToHost);
    // if (err != hipSuccess) {
    //   fprintf(stderr,
    //           "Failed to copy output from device to host (error code %s)!\n",
    //           hipGetErrorString(err));
    //   exit(EXIT_FAILURE);
    // }
    // for (size_t i = 0; i < n_graphs; i++) {
    //   printf(" graph[%zu] is connected? %zu\n ", i, host_output[i]);
    // }

    printf("Remaining %zu\n", n_graphs - offset);
    // On the CPU take the minimum value for this chunk and compare it with the
    // known minimum (if no minimum is known - i.e. cost=0 - then this will
    // always pick `curr_min`)
    acc = min_fn(acc, curr_min);
  }
  hipFree(dev_src_list);
  hipFree(dev_dst_list);
  hipFree(output);

  printf("min graph[%zu] has cost %zub\n", acc.graph, acc.cost);
  // Use the graph-id to find the set of edges that formed the minimum cost
  // solution
  printf("  graph[%zu] = [", acc.graph);
  size_t g[MAX_EDGES];
  combination_at_idx(acc.graph, g, 0, n_total_edges, n_edges);
  for (size_t eid = 0; eid < n_edges; eid++) {
    printf(" (%zu, %zu)", srcs[g[eid]], dsts[g[eid]]);
  }
  printf(" ]\n");
}
